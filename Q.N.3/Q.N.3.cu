#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
/***************************************************************************************
Demonstrates how to crack an encrypted password using a CUDA. Works on passwords that consist only of 2 uppercase
letters and a 2 digit integer.

Compile with:
 nvcc Q.N.3.cu -o Q.N.3
 ./Q.N.3
 
//Password Cracking using  
*****************************************************************************************/

//__global__ --> GPU function which can be launched by many blocks and threads
//__device__ --> GPU function or variables
//__host__ --> CPU function or variables


int calculate_time(struct timespec *start, struct timespec *end,
                   long long int *diff)
{
  long long int in_sec = end->tv_sec - start->tv_sec;
  long long int in_nano = end->tv_nsec - start->tv_nsec;
  if (in_nano < 0)
  {
    in_sec--;
    in_nano += 1000000000;
  }
  *diff = in_sec * 1000000000 + in_nano;
  return !(*diff > 0);
}


// where r = rawPassword and n = newPassword generated password
__device__ char* encryptDecrypt(char* r){

	char * n = (char *) malloc(sizeof(char) * 11);

	n[0] = r[0] + 2;
	n[1] = r[0] - 2;
	n[2] = r[0] + 1;
	n[3] = r[1] + 3;
	n[4] = r[1] - 3;
	n[5] = r[1] - 1;
	n[6] = r[2] + 2;
	n[7] = r[2] - 2;
	n[8] = r[3] + 4;
	n[9] = r[3] - 4;
	n[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ //checking all upper case letter limits
			if(n[i] > 122){
				n[i] = (n[i] - 122) + 97;
			}else if(n[i] < 97){
				n[i] = (97 - n[i]) + 97;
			}
		}else{ //checking number section
			if(n[i] > 57){
				n[i] = (n[i] - 57) + 48;
			}else if(n[i] < 48){
				n[i] = (48 - n[i]) + 48;
			}
		}
	}
	return n;
}

__global__ void crack(char * alphabet, char * numbers, char * password){

  char matchedPassword[4];

  matchedPassword[0] = alphabet[blockIdx.x];
  matchedPassword[1] = alphabet[blockIdx.y];

  matchedPassword[2] = numbers[threadIdx.x];
  matchedPassword[3] = numbers[threadIdx.y];

//firstLetter - 'A' - 'Z' (26 characters)
//secondLetter - 'A' - 'Z' (26 characters)
//firstNum - '0' - '9' (10 characters)
//secondNum - '0' - '9' (10 characters)

//Idx --> gives current index of the block or thread



const char* encryptedPassword = "koleki6253"; //UZ41
const char* search = encryptDecrypt(matchedPassword);
int Pass_match = 0;
while (*encryptedPassword != '\0' || *search != '\0') {
	if (*encryptedPassword == *search) {
		encryptedPassword++;
		search++;
	} else if ((*encryptedPassword == '\0' && *search != '\0') || (*encryptedPassword != '\0' && *search == '\0') || *encryptedPassword != *search) {
		Pass_match = 1;
	
		break;
	}
}
if (Pass_match == 0) {
	password = matchedPassword;
	printf("Password Found: %c%c%c%c \n", matchedPassword[0],matchedPassword[1],matchedPassword[2],matchedPassword[3]);
}


}

int main(int argc, char ** argv){

	hipError_t error;
	char * h_passwordfound;

	char h_Letters[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
	char h_Digits[26] = {'0','1','2','3','4','5','6','7','8','9'};

	char * d_Letters;
	error = hipMalloc( (void**) &d_Letters, sizeof(char) * 26); 
	if(error){
		fprintf(stderr, "hipMalloc on d_Letter return %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}
	error = hipMemcpy(d_Letters, h_Letters, sizeof(char) * 26, hipMemcpyHostToDevice);
	if(error){
		fprintf(stderr, "hipMemcpy to d_Letter return %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}

	char * d_Digits;
	error = hipMalloc( (void**) &d_Digits, sizeof(char) * 26); 
	if(error){
		fprintf(stderr, "hipMalloc on d_Digits return %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}
	error = hipMemcpy(d_Digits, h_Digits, sizeof(char) * 26, hipMemcpyHostToDevice);
	if(error){
		fprintf(stderr, "hipMemcpy to d_Digits return %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}

	char* d_password;
	error = hipMalloc( (void**) &d_password, sizeof(char) * 4); 
	if(error){
		fprintf(stderr, "hipMalloc on d_password return %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}
	
	struct timespec start, end;
    long long int time_used;

    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    

    crack<<< dim3(26,26,1), dim3(10,10,1) >>>( d_Letters, d_Digits ,d_password);
	error = hipGetLastError();
	if(error){
		fprintf(stderr, "Kernal launch returned %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}
    hipDeviceSynchronize();
    error = hipMemcpy(h_passwordfound, d_password, sizeof(char) * 4, hipMemcpyDeviceToHost);
	if(error){
		fprintf(stderr, "hipMemcpy to d_Digits return %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    calculate_time(&start, &end, &time_used);
    printf("Time taken: %f seconds OR %lld Nano Seconds\n", (time_used / 1.0e9), (time_used));


	error = hipFree(d_Letters);
	if(error){
		fprintf(stderr, "hipFree on d_Letters returned %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}
	error = hipFree(d_Digits);
	if(error){
		fprintf(stderr, "hipFree on d_Digits returned %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}
	error = hipFree(d_password);
	if(error){
		fprintf(stderr, "hipFree on d_password returned %d %s\n",error,hipGetErrorString(error));
		exit(1);
	}

	printf("Password found %s\n",h_passwordfound);


	return 0;
}
